#include "hip/hip_runtime.h"
/* Render the Mandelbrot set using Orbit Traps */
#include <stdlib.h>
#include <stdio.h>
#include <malloc.h>
#include <tiffio.h>
#include <assert.h>

/* CUDA_N is the resolution of the output image (size CUDA_N x CUDA_N) */
#define CUDA_N 16000

/* 8-bit red, green, and blue channels */
typedef struct {
	unsigned char r, g, b;
} pixel;

typedef struct {
	double a_r, a_g, a_b;
	double b_r, b_g, b_b;
	double c_r, c_g, c_b;
	double d_r, d_g, d_b;
} palette;

typedef struct {
	pixel *d_pixels;
	pixel *h_pixels;
	palette *d_palette;
	palette *h_palette;
	char *outfile;
	char *palfile;
	double esc_radius;
	int counter_max;
	double x, y, ref_x, ref_y;
	double a, b, c;
	double width;
	int linedist;
} fractal;

void
write_to_tiff (fractal *fract)
{
  int row, col, idx;
  TIFF *output;
  char *raster;
  pixel *img = (*fract).h_pixels;
  printf("Writing to file.\n");
  /* Open the output image */
  if ((output = TIFFOpen (fract->outfile, "w")) == NULL)
    {
      fprintf (stderr, "Could not open outgoing image.\n");
      exit (EXIT_FAILURE);
    }

  /* malloc space for the image lines */
  raster = (char*) malloc (CUDA_N * 3 * sizeof (char));
  if (raster == NULL)
    {
      printf ("malloc() failed in write_to_tiff.\n");
      exit (EXIT_FAILURE);
    }

  /* Write the tiff tags to the file */

  TIFFSetField (output, TIFFTAG_IMAGEWIDTH, CUDA_N);
  TIFFSetField (output, TIFFTAG_IMAGELENGTH, CUDA_N);
  TIFFSetField (output, TIFFTAG_COMPRESSION, COMPRESSION_DEFLATE);
  TIFFSetField (output, TIFFTAG_PLANARCONFIG, PLANARCONFIG_CONTIG);
  TIFFSetField (output, TIFFTAG_PHOTOMETRIC, PHOTOMETRIC_RGB);
  TIFFSetField (output, TIFFTAG_BITSPERSAMPLE, 8);
  TIFFSetField (output, TIFFTAG_SAMPLESPERPIXEL, 3);

  printf("Wrote image file tags.\n");

   for (row = 0; row < CUDA_N; row++)
    {
      for (col = 0; col < CUDA_N; col++)
	{
	  idx = row*CUDA_N + (CUDA_N - col);
	  raster[col*3] =   img[idx].r;
	  raster[col*3+1] = img[idx].g;
	  raster[col*3+2] = img[idx].b;
	}
      if (TIFFWriteScanline (output, raster, row, CUDA_N * 3) != 1)
	{
	  fprintf (stderr, "Could not write image\n");
	  exit (EXIT_FAILURE);
	}
    }

  free (raster);
  /* close the file */
  TIFFClose (output);
}

/* color(t) = a + b * cos[2pi(c*t+d)] */
__device__ void
color_pxl(double t, palette *pal, double *r_out, double *g_out, double *b_out)
{
	*r_out = 255. * (pal->a_r + pal->b_r * cos(M_PI * 2. * (pal->c_r * t + pal->d_r)));
	*g_out = 255. * (pal->a_g + pal->b_g * cos(M_PI * 2. * (pal->c_g * t + pal->d_g)));
	*b_out = 255. * (pal->a_b + pal->b_b * cos(M_PI * 2. * (pal->c_b * t + pal->d_b)));
}


/* distance between (x1, y1) and (x2, y2) */
__device__ double
point_dist(double x1, double x2, double y1, double y2)
{
	return sqrt((x2-x1)*(x2-x1)+(y2-y1)*(y2-y1));
}

/* distance between (x0, y0) and line (ax+by+c=0) */
__device__ double
line_dist(double x0, double y0, double a, double b, double c)
{
	double d, n;
	d = sqrt(a*a+b*b);
	n = abs(a*x0+b*y0+c);
	return n/d;
}

__global__ void 
render(pixel *pxls, 
float xmin, float xmax, float ymin, float ymax, 
double esc, int count_max, 
double xref, double yref,
double a, double b, double c, int linedist, 
palette *pal)
{
	int i, j, idx;
	float x1, y1, x2, y2, xtmp;
	int counter = 0;
	double dist = 1e9;

	double r_out, g_out, b_out;

	/* compute x (i) and y (j) index from Block and Thread */
	i = blockIdx.x * blockDim.x + threadIdx.x;
	j = blockIdx.y * blockDim.y + threadIdx.y;
	if(i >= CUDA_N || j >= CUDA_N) return; /* verify inbounds of image */ 	

	/* find x and y cartesian points for pixel */
	x1 = xmax - ( ((float) i / (float) CUDA_N) * (xmax - xmin) );
	y1 = ymax - ( ((float) j / (float) CUDA_N) * (ymax - ymin) );
	
	x2 = x1;
	y2 = y1;
	while( ( (x1*x1 + y1*y1) < esc ) && counter < count_max )
	{
		xtmp = x1 * x1 - y1 * y1 + x2;
		y1 = 2. * x1 * y1 + y2;
		x1 = xtmp;
		counter++;
		
		dist = min(dist, 
		linedist == 0 ?	point_dist(x1,xref,y1,yref) : line_dist(x1, y1, a, b, c));
	}
	idx = i + j*CUDA_N;
	color_pxl(dist, pal, &r_out, &g_out, &b_out);
	pxls[idx].r = (char) r_out;
	pxls[idx].g = (char) g_out;
	pxls[idx].b = (char) b_out;
}

/* initialize the color palette with user inputs
 * or a default state if no input is provided. */
void pal_init(palette *pal, char *infile)
{
	FILE *palette;

	if(infile == NULL) {
		/* a nice light blue default */
		pal->a_r = 0.39;
		pal->a_g = 0.55;
		pal->a_b = 0.5;

		pal->b_r = 0.55;
		pal->b_g = 0.26;
		pal->b_b = 0.68;

		pal->c_r = 0.5;
		pal->c_g = 1.5;
		pal->c_b = 0.0;

		pal->d_r = 0.26;
		pal->d_g = 0.11;
		pal->d_b = 0.24;
	} else {
	  if ((palette = fopen(infile, "r")) == NULL)
	  {
	   printf ("Error reading input file %s.\n", infile);
	   exit (EXIT_FAILURE);
	  }
		/* WARNING -- poor checks for malformed input here. */
		assert(fscanf (palette, "%lf %lf %lf\n", &(pal->a_r), &(pal->a_g), &(pal->a_b)) != EOF);
		assert(fscanf (palette, "%lf %lf %lf\n", &(pal->b_r), &(pal->b_g), &(pal->b_b)) != EOF);
		assert(fscanf (palette, "%lf %lf %lf\n", &(pal->c_r), &(pal->c_g), &(pal->c_b)) != EOF);
		assert(fscanf (palette, "%lf %lf %lf\n", &(pal->d_r), &(pal->d_g), &(pal->d_b)) != EOF);
		(void) fclose (palette);
	} /* end else */
} /* end pal_init */

void
print_usage ()
{
  /* print program use */

  printf ("Render the Mandelbrot set using Orbit Traps.\n\n");
  printf ("mandel usage:\n");
  printf ("mandel [-options ...]\n\n");
  printf ("options include:\n");

  printf ("\t-h\t\t\tprint this screen\n");
  printf ("\t-f NAME\t\t\toutput file to use (i.e. /tmp/mandel.tif)\n");
  printf ("\t-p NAME\t\t\tfile to use for color palette\n");
  printf ("\t-x #.###...#\t\tcenter X coordinate of image\n");
  printf ("\t-y #.###...#\t\tcenter Y coordinate of image\n");
  printf ("\t-rx #.###...#\t\tX coordinate for distance reference\n");
  printf ("\t-ry #.###...#\t\tY coordinate for distance reference\n");
  printf ("\t-L\t\t\tuse the line equation for orbit trap instead of a point\n");
  printf ("\t-a #.###...#\t\tA parameter of reference line in form Ax + By + C = 0\n");
  printf ("\t-b #.###...#\t\tB parameter of reference line in form Ax + By + C = 0\n");
  printf ("\t-c #.###...#\t\tC parameter of reference line in form Ax + By + C = 0\n");
  printf ("\t-w ##.#\t\t\twidth of image (x and y +/- width)\n");
  printf ("\t-m ####\t\t\tmax iterations to compute\n");
  printf ("\t-e ##.#\t\t\tescape radius\n");


}

void
parse_args (int argc, char **argv, fractal * mandel)
{
  int i = 1;
  while (i < argc)
    {
		if (!strcmp (argv[i], "-h"))
		{
	  	print_usage ();
	  	exit (EXIT_SUCCESS);
		}
		else if (!strcmp (argv[i], "-f"))
		{
	  	mandel->outfile = argv[i + 1];
	 	i += 2;
		}
		else if (!strcmp (argv[i], "-p"))
		{
	  	mandel->palfile = argv[i + 1];
	  	i += 2;
		}
		else if (!strcmp (argv[i], "-x"))
		{
	  	mandel->x = (double) atof(argv[i + 1]);
	 	i += 2;
		}
		else if (!strcmp (argv[i], "-y"))
		{
	  	mandel->y = (double) atof(argv[i + 1]);
	 	i += 2;
		}
		else if (!strcmp (argv[i], "-rx"))
		{
	  	mandel->ref_x = (double) atof(argv[i + 1]);
	 	i += 2;
		}
		else if (!strcmp (argv[i], "-ry"))
		{
	  	mandel->ref_y = (double) atof(argv[i + 1]);
	 	i += 2;
		}
		else if (!strcmp (argv[i], "-a"))
		{
	  	mandel->a = (double) atof(argv[i + 1]);
	 	i += 2;
		}
		else if (!strcmp (argv[i], "-b"))
		{
	  	mandel->b = (double) atof(argv[i + 1]);
	 	i += 2;
		}
		else if (!strcmp (argv[i], "-c"))
		{
	  	mandel->c = (double) atof(argv[i + 1]);
	 	i += 2;
		}
		else if (!strcmp (argv[i], "-w"))
		{
	  	mandel->width = (double) atof(argv[i + 1]);
	 	i += 2;
		}
		else if (!strcmp (argv[i], "-m"))
		{
	  	mandel->counter_max = atoi(argv[i + 1]);
	 	i += 2;
		}
		else if (!strcmp (argv[i], "-L"))
		{
	  	mandel->linedist = 1;
	 	i += 1;
		}
		else if (!strcmp (argv[i], "-e"))
		{
	  	mandel->esc_radius = atof(argv[i + 1]);
	 	i += 2;
		}
		else
		{
	 	print_usage ();
	  	exit (EXIT_FAILURE);
		}	
	}
}	

int main(int argc, char **argv)
{
	fractal mandel;
	mandel.d_pixels = NULL;
	mandel.h_pixels = NULL;
	mandel.d_palette = NULL;
	mandel.h_palette = NULL;
	mandel.outfile   = (char *) "/tmp/mandel.tif"; /* default */
	mandel.palfile   = NULL;
	mandel.esc_radius= 2e5;
	mandel.counter_max = 350;
	mandel.x = 0.0;
	mandel.y = 0.0;
	mandel.ref_x = 0.0;
	mandel.ref_y = 0.0;
	mandel.width = 2.5;
	mandel.a = 1.0;
	mandel.b = -1.0;
	mandel.c = 0.0;
	mandel.linedist = 0;
	hipError_t err;

	/* process input arguments */
	parse_args(argc, argv, &mandel);
	/* sanity check */
	if(mandel.linedist == 1 && (mandel.a == 0.0 && mandel.b == 0.0)) {
		printf("Illegal configuration.  A and B cannot both be set to zero.\n");
		exit(EXIT_FAILURE);
	}

	/* HOST buffer for color palette */
	mandel.h_palette = (palette*) malloc(sizeof(palette));
	if(mandel.h_palette == NULL) {
		printf("malloc() failed in main.\n");
		exit(EXIT_FAILURE);
	}

	/* Initialize the palette */
	pal_init(mandel.h_palette, mandel.palfile);	

	/* assign a CUDA memory buffer for the fractal rendering */
	err = hipMalloc(&(mandel.d_pixels), CUDA_N*CUDA_N*sizeof(pixel));
	if(err != hipSuccess) {
		printf("%s\n", hipGetErrorString(err));		
		exit(EXIT_FAILURE);
	}

	/* assign a CUDA memory buffer for the color palette */
	err = hipMalloc(&(mandel.d_palette), sizeof(palette));
	if(err != hipSuccess) {
		printf("%s\n", hipGetErrorString(err));		
		exit(EXIT_FAILURE);
	}	
	
	printf("Allocated CUDA device memory.\n");

	/* setup block sizes to allow for rendering in min number of blocks */
	dim3 threadsPerBlock(16, 16);
	dim3 numBlocks(CUDA_N / threadsPerBlock.x, CUDA_N / threadsPerBlock.y);

	/* copy palette to device */
	/* copy the buffer from HOST to DEVICE */
	err = hipMemcpy(mandel.d_palette, mandel.h_palette, sizeof(palette), hipMemcpyHostToDevice);
	        if(err != hipSuccess) {
                printf("%s\n", hipGetErrorString(err));
                exit(EXIT_FAILURE);
        }

	/* dispatch the CUDA process */
	render<<<numBlocks, threadsPerBlock>>>(mandel.d_pixels,
	mandel.x-mandel.width, mandel.x+mandel.width, mandel.y-mandel.width, mandel.y+mandel.width,
	mandel.esc_radius, mandel.counter_max,
	mandel.ref_x, mandel.ref_y,
	mandel.a, mandel.b, mandel.c, mandel.linedist,
	mandel.d_palette);
	printf("Completed render.\n");

	/* HOST buffer for completed render */
	mandel.h_pixels = (pixel*) malloc(CUDA_N*CUDA_N*sizeof(pixel));
	if(mandel.h_pixels == NULL) {
		printf("malloc() failed in main.\n");
		exit(EXIT_FAILURE);
	}

	/* copy the buffer from DEVICE to HOST */
	err = hipMemcpy(mandel.h_pixels, mandel.d_pixels, CUDA_N*CUDA_N*sizeof(pixel), hipMemcpyDeviceToHost);
	        if(err != hipSuccess) {
                printf("%s\n", hipGetErrorString(err));
                exit(EXIT_FAILURE);
        }
	printf("Mem copy complete.\n");
	/* then free the DEVICE memory */
	hipFree(mandel.d_pixels);
	hipFree(mandel.d_palette);
	printf("Freed CUDA memory.\n");
	/* then write the buffer to file */
	write_to_tiff(&mandel);
	/* and free the buffer */
	printf("Wrote to file.\n");
	free (mandel.h_pixels);
	free (mandel.h_palette);
	return 0;
}
	
